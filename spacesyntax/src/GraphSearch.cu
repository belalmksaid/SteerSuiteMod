#include "hip/hip_runtime.h"
#define BLOCK_SIZE 256
#define MAX_DEPTH 10
#include <math.h>
#include <iostream>
#include "Structs.h"
#include "hip/hip_runtime.h"
#include <>
#include "GraphSearch.h"
#include <time.h>
#include <vector>

using namespace std;

bool selectBestGPU()
{
	int numDevices;
	hipError_t cudaResultCode = hipGetDeviceCount(&numDevices);
	if (cudaResultCode != hipSuccess){
		return false;
	}

	int maxMultiProc = 0;
	for (int i = 0; i < numDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		if (prop.multiProcessorCount > maxMultiProc) {
			maxMultiProc = prop.multiProcessorCount;
			hipSetDevice(i);
		}
	}

	return true;
}

__device__ unsigned return_thread_index()
{
	unsigned thread_num_in_block = threadIdx.x + (threadIdx.y * blockDim.x);
	unsigned block_num_in_grid = blockIdx.x + (blockIdx.y * gridDim.x);
	return (block_num_in_grid * blockDim.x * blockDim.y) + thread_num_in_block;
}

__device__ float solve_line_z(float xTarget, float x1, float z1, float tangent)
{
	return tangent*(xTarget - x1) + z1;
}

__device__ float solve_line_x(float zTarget, float x1, float z1, float tangent)
{
	return (zTarget - z1) / tangent + x1;
}


__device__ void rotate_axis(float& x1r, float& z1r, float tilt, float x1, float z1)
{
	float sinVal, cosVal;
	sinVal = sinf(tilt);
	cosVal = cosf(tilt);
	z1r = cosVal*z1 - sinVal*x1;
	x1r = sinVal*z1 + cosVal*x1;
}

__device__ int ray_cast(Point_Struct p1, Point_Struct p2, Obstacle_Struct* obstacles, int numObstacles)
{
	for (int i = 0; i < numObstacles; i++)
	{
		Obstacle_Struct obstacle = obstacles[i];
		float x1, z1, x2, z2, target;
		if (obstacle.type == SHADOW_OBSTACLE)
			continue;

		x1 = obstacle.pMin.x;
		z1 = obstacle.pMin.z;
		x2 = obstacle.pMax.x;
		z2 = obstacle.pMax.z;

		// Do axis rotation to get axis aligned obstacle
		float tilt, x1r, z1r, x2r, z2r, px1r, pz1r, px2r, pz2r, tangent;
		tilt = obstacle.tiltDegree;
		rotate_axis(x1r, z1r, tilt, x1, z1);
		rotate_axis(x2r, z2r, tilt, x2, z2);
		rotate_axis(px1r, pz1r, tilt, p1.x, p1.z);
		rotate_axis(px2r, pz2r, tilt, p2.x, p2.z);
		tangent = (pz2r - pz1r) / (px2r - px1r);

		float xpMin = (px1r < px2r) ? px1r : px2r;
		float xpMax = (px1r > px2r) ? px1r : px2r;
		float zpMin = (pz1r < pz2r) ? pz1r : pz2r;
		float zpMax = (pz1r > pz2r) ? pz1r : pz2r;

		if (x1r > x2r)
		{
			float tmp = x1r;
			x1r = x2r;
			x2r = tmp;
		}
		if (z1r > z2r)
		{
			float tmp = z1r;
			z1r = z2r;
			z2r = tmp;
		}

		// check if the obstacle is within points
		if (xpMin > x2r || xpMax < x1r || zpMin > z2r || zpMax < z1r)
			continue;

		// check if any of the points are inside the obstacle
		if (px1r >= x1r && px1r <= x2r && pz1r >= z1r && pz1r <= z2r)
			return obstacle.type;
		if (px2r >= x1r && px2r <= x2r && pz2r >= z1r && pz2r <= z2r)
			return obstacle.type;

		if (x1r >= xpMin && x1r <= xpMax)
		{
			target = solve_line_z(x1r, px1r, pz1r, tangent);
			if (target >= z1r && target <= z2r)
				if (target >= zpMin && target <= zpMax)
					return obstacle.type;
		}

		if (x2r >= xpMin && x2r <= xpMax)
		{
			target = solve_line_z(x2r, px1r, pz1r, tangent);
			if (target >= z1r && target <= z2r)
				if (target >= zpMin && target <= zpMax)
					return obstacle.type;
		}
		if (z1r >= zpMin && z1r <= zpMax)
		{
			target = solve_line_x(z1r, px1r, pz1r, tangent);
			if (target >= x1r && target <= x2r)
				if (target >= xpMin && target <= xpMax)
					return obstacle.type;
		}

	}
	return SHADOW_OBSTACLE;
}

__device__ bool hasLineOfSight(Point_Struct point1, Point_Struct point2, Obstacle_Struct* obstacles, int numObstacles)
{
	if (ray_cast(point1, point2, obstacles, numObstacles) == BLOCK_OBSTACLE)
		return false;
	else
		return true;
}


__global__ void cuda_generate_graph(Point_Struct* grid, int* adjMatrix_ptr, int totalNodes, Obstacle_Struct* obstacles, int numObstacles)
{
	int threadIndex = return_thread_index();
	Point_Struct pt1 = grid[threadIndex];
	if (threadIndex < totalNodes)
	{
		for (int i = 0; i < totalNodes; i++)
		{
			Point_Struct pt2 = grid[i];
			if (hasLineOfSight(pt1, pt2, obstacles, numObstacles))
			{
				int index = totalNodes * threadIndex + i;
				adjMatrix_ptr[index] = 1;
			}
		}
	}
}

__global__ void cuda_generate_graph_mv1(Point_Struct* grid, bool* adjMatrix_ptr, int totalNodes, Obstacle_Struct* obstacles, int numObstacles)
{
	unsigned threadIndex = return_thread_index();
	unsigned threadIndexY = threadIndex / totalNodes;
	unsigned threadIndexX = threadIndex % totalNodes;
	Point_Struct pt1 = grid[threadIndexY];
	Point_Struct pt2 = grid[threadIndexX];
	if (threadIndexX < totalNodes && threadIndexY <= threadIndexX) //the first condition is actually always true, Y check makes sense
	{
		if (hasLineOfSight(pt1, pt2, obstacles, numObstacles))
		{
			unsigned index = totalNodes * threadIndexY + threadIndexX;
			unsigned index2 = totalNodes * threadIndexX + threadIndexY;
			adjMatrix_ptr[index] = true; //maybe use symmetry
			adjMatrix_ptr[index2] = true;
		}
		else
		{
			unsigned index = totalNodes * threadIndexY + threadIndexX;
			unsigned index2 = totalNodes * threadIndexX + threadIndexY;
			adjMatrix_ptr[index] = false; //maybe use symmetry
			adjMatrix_ptr[index2] = false;
		}
	}
}

__global__ void cuda_sequence_list(bool* frontMat, int* seqFront, int totalNodes)
{
	unsigned tid, count, ntree;
	count = 0;
	tid = return_thread_index();
	if (tid >= totalNodes)
		return;
	ntree = tid % totalNodes;
	for (unsigned s = 0; s < totalNodes; s++)
	{
		if (frontMat[s + ntree * totalNodes])
		{
			seqFront[count + ntree * totalNodes] = s;
			count++;
		}
	}
}

__global__ void cuda_expand_forest(bool* parentMat, bool* childMat, int* seqFront, int* depthMat, Point_Struct* nodeList, bool* adjMatrix, int depth, int totalNodes)
{
	int tid, ntree, idX, idY;
	ntree = blockIdx.y;
	idX = threadIdx.x + (threadIdx.y * blockDim.x) + (blockIdx.x * BLOCK_SIZE);
	tid = threadIdx.x + (threadIdx.y * blockDim.x);
	if (ntree >= totalNodes || idX >= totalNodes)
		return;

	//extern __shared__ int seqf[];
	__shared__ int frontDepth;
	if (tid == 0)
	{
		if (depth == 0)
			frontDepth = 1;
		else
			frontDepth = depthMat[depth - 1 + ntree * MAX_DEPTH];
	}
	/*
	for (unsigned s = 0; s < totalNodes; s += BLOCK_SIZE)
	{
	if (tid + s < totalNodes)
	{
	seqf[tid + s] = seqFront[tid + s + ntree * totalNodes];
	}
	}*/
	__syncthreads();

	if (!nodeList[ntree].isQ || !nodeList[idX].isRef)
		return;
	for (unsigned s = 0; s < frontDepth; s++)
	{
		if (parentMat[idX + ntree * totalNodes])
			return;
		//idY = seqf[s];
		idY = seqFront[s + ntree * totalNodes];
		if (idY < 0)
			return;
		if (adjMatrix[idX + idY * totalNodes])
		{
			childMat[idX + ntree * totalNodes] = true;
			parentMat[idX + ntree * totalNodes] = true;
		}
	}
}

__global__ void cuda_init_forest(bool* parentMat, bool* childMat, bool* frontMat, int* seqFront, int* depthMat, int totalNodes)
{
	unsigned tid, idX, idY;
	tid = return_thread_index();
	idX = tid % totalNodes;
	idY = tid / totalNodes;

	if (idY >= totalNodes)
		return;

	childMat[idX + idY * totalNodes] = false;
	seqFront[idX + idY * totalNodes] = -1;
	if (idX == idY)
	{
		frontMat[idX + idY * totalNodes] = true;
		parentMat[idX + idY * totalNodes] = true;
	}
	else
	{
		frontMat[idX + idY * totalNodes] = false;
		parentMat[idX + idY * totalNodes] = false;
	}

	if (idX < MAX_DEPTH)
	{
		depthMat[idX + idY * MAX_DEPTH] = 0;
	}
}

__global__ void cuda_update_forest(bool* parentMat, bool* childMat, bool* frontMat, int* seqFront, int* depthMat, int depth, int totalNodes)
{
	unsigned tid, idX, idY;
	tid = return_thread_index();
	idX = tid % totalNodes;
	idY = tid / totalNodes;
	extern __shared__ int sumVal[];

	if (idY >= totalNodes)
		return;

	// Init shared memory (only where needed)
	if (sumVal[idY] != 0)
		sumVal[idY] = 0;
	__syncthreads();


	if (childMat[idX + idY * totalNodes])
		atomicAdd(&sumVal[idY], 1);
	__syncthreads();

	if ((threadIdx.x == 0 || idX == 0) && sumVal[idY] > 0)
		atomicAdd(&depthMat[depth + idY * MAX_DEPTH], sumVal[idY]);

	//if (childMat[idX + idY * totalNodes])
	//	atomicAdd(&depthMat[depth + idY * MAX_DEPTH], 1);
	frontMat[idX + idY * totalNodes] = childMat[idX + idY * totalNodes];
	childMat[idX + idY * totalNodes] = false;
	seqFront[idX + idY * totalNodes] = -1;
}

__global__ void cuda_extract_forest(int* dresults, float* eresults, int* depthMat, Point_Struct* nodeList, int totalNodes)
{
	unsigned depth, ntree;
	depth = threadIdx.x;
	ntree = blockIdx.x;
	__shared__ int dval, total;
	__shared__ float eval;
	if (depth == 0)
	{
		dval = 0;
		eval = 0;
		total = 0;
	}
	__syncthreads();

	if (depthMat[depth + ntree*MAX_DEPTH] > 0 && nodeList[ntree].isQ)
	{
		atomicAdd(&total, depthMat[depth + ntree*MAX_DEPTH]);
		atomicAdd(&dval, 1);
	}
	__syncthreads();

	if (depthMat[depth + ntree*MAX_DEPTH] > 0 && nodeList[ntree].isQ)
	{
		float p;
		p = (float)depthMat[depth + ntree*MAX_DEPTH] / (float)total;
		atomicAdd(&eval, -p*log2(p));
	}
	__syncthreads();

	if (depth == 0)
		dresults[ntree] = dval;
	else if (depth == 1)
		eresults[ntree] = eval;

}

__global__ void cuda_calc_degree(int* kresults, bool* adjMat, Point_Struct* nodeList, int totalNodes)
{
	unsigned tid, idX, idY;
	tid = return_thread_index();
	idX = tid % totalNodes;
	idY = tid / totalNodes;
	extern __shared__ int sumVal1[];

	if (idY >= totalNodes)
		return;
	if (idY == 0)
	{
		kresults[idX] = 0;
	}

	// Init shared memory (only where needed)
	if (sumVal1[idY] != 0)
		sumVal1[idY] = 0;
	__syncthreads();

	if (adjMat[idX + idY * totalNodes] && nodeList[idX].isRef)
		atomicAdd(&sumVal1[idY], 1);
	__syncthreads();

	if (sumVal1[idY] > 0 && nodeList[idY].isQ && ((threadIdx.x + threadIdx.y) == 0 || idX == 0))
		atomicAdd(&kresults[idY], sumVal1[idY]);
}

__global__ void reduced_add_int(int* g_idata, int* g_odata, int totalNodes)
{
	extern __shared__ int sdata[];

	// Read chuncks into shared memory of each block
	unsigned tid = threadIdx.x + threadIdx.y * blockDim.x;
	unsigned i = return_thread_index();
	if (i < totalNodes)
		sdata[tid] = g_idata[i];
	else
		sdata[tid] = 0;
	__syncthreads();

	// Do reduction
	for (unsigned s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// Write resutls
	if (tid == 0)
		atomicAdd(g_odata, sdata[0]);
}

__global__ void reduced_add_float(float* g_idata, float* g_odata, int totalNodes)
{
	extern __shared__ float sdata1[];

	// Read chuncks into shared memory of each block
	unsigned tid = threadIdx.x + threadIdx.y * blockDim.x;
	unsigned i = return_thread_index();
	if (i < totalNodes)
		sdata1[tid] = g_idata[i];
	else
		sdata1[tid] = 0;
	__syncthreads();

	// Do reduction
	for (unsigned s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata1[tid] += sdata1[tid + s];
		}
		__syncthreads();
	}

	// Write resutls
	if (tid == 0)
		atomicAdd(g_odata, sdata1[0]);
}

/**************GPU VISIBILITY GRAPH********************/

gpuVG::~gpuVG()
{
	clear();
}

void gpuVG::clear()
{
	//Free memory
	hipFree(_adjMatrix);
	hipFree(_nodeList);
	hipFree(_obsList);
	hipFree(_depthMat);
	hipFree(_parentMat);
	hipFree(_childMat);
	hipFree(_frontMat);
	hipFree(_kresults);
	hipFree(_dresults);
	hipFree(_eresults);
	hipFree(_output);
	_isInit = false;
}

void gpuVG::init(vector<Point_Struct> nodeList, unsigned obsCount)
{
	if (_isInit)
		return;

	int totalNodes = nodeList.size();
	// Visibility Grid
	hipMalloc((void**)&_nodeList, totalNodes*sizeof(Point_Struct));
	hipMemcpy(_nodeList, &nodeList[0], totalNodes*sizeof(Point_Struct), hipMemcpyHostToDevice);

	//Initialize adjacency matrix to 0
	hipMalloc((void**)&_adjMatrix, totalNodes*totalNodes*sizeof(bool));
	hipMalloc((void**)&_parentMat, totalNodes*totalNodes*sizeof(bool));
	hipMalloc((void**)&_childMat, totalNodes*totalNodes*sizeof(bool));
	hipMalloc((void**)&_frontMat, totalNodes*totalNodes*sizeof(bool));
	hipMalloc((void**)&_seqFront, totalNodes*totalNodes*sizeof(int));
	//hipMemset(_adjMatrix, false, totalNodes*totalNodes*sizeof(bool));

	// Initialize depth container
	hipMalloc((void**)&_depthMat, totalNodes*MAX_DEPTH*sizeof(int));

	// Init results
	hipMalloc((void**)&_kresults, totalNodes*sizeof(int));
	hipMalloc((void**)&_dresults, totalNodes*sizeof(int));
	hipMalloc((void**)&_eresults, totalNodes*sizeof(float));
	hipMalloc((void**)&_output, 3*sizeof(int));

	// MEMORY LEAK DANGER IN CASE OBSTABLE NUMBER IS NOT CONSTANT
	hipMalloc((void**)&_obsList, obsCount*sizeof(Obstacle_Struct));

	_totalNodes = totalNodes;
	_isInit = true;
}

void gpuVG::generate_graph(vector<Obstacle_Struct> obstacles)
{
	// Setup block and grid size
	int blockLength = sqrt((double)BLOCK_SIZE);
	int gridLength = ceil(sqrt((double)_totalNodes*_totalNodes / ((double)BLOCK_SIZE)));
	dim3 threads(blockLength, blockLength, 1);
	dim3 blocks(gridLength, gridLength, 1);

	//Set obstacles to GPU memory
	Obstacle_Struct* obstacle_ptr = &obstacles[0];
	int numObstacles = obstacles.size();
	hipMemcpy(_obsList, obstacle_ptr, obstacles.size()*sizeof(Obstacle_Struct), hipMemcpyHostToDevice);

	// Run kernel
	cuda_generate_graph_mv1 << < blocks, threads >> >(_nodeList, _adjMatrix, _totalNodes, _obsList, obstacles.size());

	// Calculate degree
	cuda_calc_degree << <blocks, threads, _totalNodes*sizeof(int) >> >(_kresults, _adjMatrix, _nodeList, _totalNodes);
}

bool* gpuVG::get_adjMatrix()
{
	//Copy memory back to host
	bool* adjMatrix_ptr = (bool*)malloc(_totalNodes*_totalNodes*sizeof(bool));
	hipMemcpy(adjMatrix_ptr, _adjMatrix, _totalNodes*_totalNodes*sizeof(bool), hipMemcpyDeviceToHost);
	return adjMatrix_ptr;
}

int* gpuVG::get_depthMatrix()
{
	//Copy memory back to host
	int* depthMatrix_ptr = (int*)malloc(_totalNodes*MAX_DEPTH*sizeof(int));
	hipMemcpy(depthMatrix_ptr, _depthMat, _totalNodes*MAX_DEPTH*sizeof(int), hipMemcpyDeviceToHost);
	return depthMatrix_ptr;
}

int* gpuVG::get_degree()
{
	int blockLength = ceil(sqrt((double)BLOCK_SIZE));
	int gridLength = ceil((double)_totalNodes / BLOCK_SIZE);
	int gridLengthX = ceil((double)_totalNodes*_totalNodes / BLOCK_SIZE);
	dim3 threads(blockLength, blockLength, 1);
	dim3 blocks(gridLength, 1, 1);
	dim3 blocks_2(gridLengthX, 1, 1);

	// Initialize output
	hipMemset(_output, 0, sizeof(int));

	// Calculate degree
	//cuda_calc_degree << <blocks_2, threads, _totalNodes*sizeof(int) >> >(_kresults, _adjMatrix, _nodeList, _totalNodes);

	// Take sum over nodes
	reduced_add_int << <blocks, threads, BLOCK_SIZE*sizeof(int) >> >(_kresults, &_output[0], _totalNodes);
	int* output = (int*)malloc(sizeof(int));
	hipMemcpy(output, _output, sizeof(int), hipMemcpyDeviceToHost);
	return output;
}

void gpuVG::generate_forest()
{
	// Setup block and grid size
	int blockLength = ceil(sqrt((double) BLOCK_SIZE));
	int gridLengthX = ceil((double) _totalNodes*_totalNodes/BLOCK_SIZE);
	int gridLengthY = ceil((double)_totalNodes / BLOCK_SIZE);
	dim3 threads(BLOCK_SIZE, 1, 1);
	dim3 blocks_2(gridLengthX, 1, 1);
	dim3 blocks_1(gridLengthY, 1, 1);
	dim3 blocks_3(gridLengthY, _totalNodes, 1);
	hipError_t err;

	// Init tree
	cuda_init_forest << < blocks_2, threads >> >(_parentMat, _childMat, _frontMat, _seqFront, _depthMat, _totalNodes);
	//int* res = (int*)malloc(_totalNodes*_totalNodes*sizeof(int));
	for (int i = 0; i < MAX_DEPTH; i++)
	{
		// Make front list
		cuda_sequence_list << < blocks_1, threads >> >(_frontMat, _seqFront, _totalNodes);

		// Run kernel
		//cuda_expand_forest << < blocks_3, threads, _totalNodes*sizeof(int) >> >(_parentMat, _childMat, _seqFront, _depthMat, _nodeList, _adjMatrix, i, _totalNodes);
		cuda_expand_forest << < blocks_3, threads >> >(_parentMat, _childMat, _seqFront, _depthMat, _nodeList, _adjMatrix, i, _totalNodes);
		
		// Update tree
		cuda_update_forest << < blocks_2, threads, _totalNodes*sizeof(int) >> >(_parentMat, _childMat, _frontMat, _seqFront, _depthMat, i, _totalNodes);
	}
	dim3 threadsD(MAX_DEPTH, 1, 1);
	dim3 blocksD(_totalNodes, 1, 1);

	// Calculate depth and entropy
	cuda_extract_forest << <blocksD, threadsD >> >(_dresults, _eresults, _depthMat, _nodeList, _totalNodes);
}

int* gpuVG::get_output()
{
	int blockLength = ceil(sqrt((double)BLOCK_SIZE));
	int gridLength = ceil((double)_totalNodes / BLOCK_SIZE);
	int gridLengthX = ceil((double)_totalNodes*_totalNodes / BLOCK_SIZE);
	dim3 threads(blockLength, blockLength, 1);
	dim3 blocks(gridLength, 1, 1);
	dim3 blocks_2(gridLengthX, 1, 1);

	// Initialize output
	hipMemset(_output, 0, 3 * sizeof(int));

	// Calculate degree
	//cuda_calc_degree << <blocks_2, threads, _totalNodes*sizeof(int) >> >(_kresults, _adjMatrix, _nodeList, _totalNodes);

	// Take sum over nodes
	reduced_add_int << <blocks, threads, BLOCK_SIZE*sizeof(int) >> >(_kresults, &_output[0], _totalNodes);
	reduced_add_int << <blocks, threads, BLOCK_SIZE*sizeof(int) >> >(_dresults, &_output[1], _totalNodes);
	reduced_add_float << <blocks, threads, BLOCK_SIZE*sizeof(float) >> >(_eresults, (float*)&_output[2], _totalNodes);
	int* output = (int*) malloc(3 * sizeof(int));
	hipMemcpy(output, _output, 3 * sizeof(int), hipMemcpyDeviceToHost);
	return output;
}

int* gpuVG::get_nodesDegree()
{
	int* output = (int*)malloc(_totalNodes * sizeof(int));
	hipMemcpy(output, _kresults, _totalNodes * sizeof(int), hipMemcpyDeviceToHost);
	return output;
}

int* gpuVG::get_nodesDepth()
{
	int* output = (int*)malloc(_totalNodes * sizeof(int));
	hipMemcpy(output, _dresults, _totalNodes * sizeof(int), hipMemcpyDeviceToHost);
	return output;
}

float* gpuVG::get_nodesEntropy()
{
	float* output = (float*)malloc(_totalNodes * sizeof(float));
	hipMemcpy(output, _eresults, _totalNodes * sizeof(float), hipMemcpyDeviceToHost);
	return output;
}
